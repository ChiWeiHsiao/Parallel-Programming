/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>    
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
__global__ void init_line(int, float*, float*);
__global__ void update (int, int, float*, float*);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
//float  values[MAXPOINTS+2], 	/* values at time t */
//       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
//       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
float *values, *oldval, *newval;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
__global__ void init_line(int __tpoints, float* __oldval, float* __newval)//(tpoints, oldval, newval)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //float x;
  if (i < __tpoints) {
    float x = (float) i / (__tpoints - 1);
    __newval[i] = __sinf(6.2831853 * x);
    __oldval[i] = __newval[i];
  }
//=============================================
  /*int i, j;
   float x, k, tmp; 
   // Calculate initial values based on sine curve 
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (6.2831853 * x);//fac = 2.0 * PI;
      k = k + 1.0;
   } 
   //Initialize old values array 
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];*/
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i) // The Effect is: newval[i] = (1.82)*values[i] - oldval[i] 
{
   float dtime, c, dx, tau, sqtau;
   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);  //=0.3
   sqtau = tau * tau; //=0.09
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);// = (2-0.18)*values[i] - oldval[i]
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(int __nsteps, int __tpoints, float* __oldval, float* __newval)  //(nsteps, tpoints, oldval, newval);
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < __tpoints) {
    float tmp_old = __oldval[i];
    float tmp_new = __newval[i];
    float next;
    for (int j = 0; j < __nsteps; j++) {
      // global endpoints 
      if ((i == 0) || (i == __tpoints - 1))
        next = 0;
      else
        //do_math, newval[i] = (1.82)*values[i] - oldval[i] 
        next = (1.82) * tmp_new - tmp_old;
      tmp_old = tmp_new;
      tmp_new = next;
    }
    __newval[i] = tmp_new;
  }
  //==============
  /* Update values for each time step 
   for (i = 1; i<= nsteps; i++) {
      // Update points along line for this time step 
      for (j = 1; j <= tpoints; j++) {
         /* global endpoints 
         if ((j == 1) || (j  == tpoints))
            newval[j] = 0.0;
         else
            do_math(j);
      }

      // Update old values with new values 
      for (j = 1; j <= tpoints; j++) {
         oldval[j] = values[j];
         values[j] = newval[j];
      }
   }*/
}
/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;
   //for (i = 1; i <= tpoints; i++) { //stat from 1
   for (i = 0; i < tpoints; i++) {  //start from 0
      printf("%6.4f ", values[i]);
      if ((i+1)%10 == 0)  //start from 0, so +1
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
//clock_t t = clock();//time
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
  /* Memory */
  values = (float*) malloc( (tpoints+256) * sizeof(float));
  hipMalloc((void**) &newval, (tpoints+256) * sizeof(float));
  hipMalloc((void**) &oldval, (tpoints+256) * sizeof(float));
  /* Invoke kernel*/
  dim3 threadsPerBlock(256);
  dim3 numOfBlocks((tpoints+256)/256);
	printf("Initializing points on the line...\n");
	init_line<<<numOfBlocks, threadsPerBlock>>>(tpoints, oldval, newval);
	printf("Updating all points for all time steps...\n");
	update<<<numOfBlocks, threadsPerBlock>>>(nsteps, tpoints, oldval, newval);
  //Move from gpu to cpu
  hipMemcpy(values, newval, (tpoints+256)*sizeof(float), hipMemcpyDeviceToHost);

	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
  //Free all memory
  hipFree(newval);
  hipFree(oldval);
  free(values);
//!!!!!記得刪掉時間!!!!!
//t = clock() - t;//time
//printf("Time : %f sec\n", ((float)t)/CLOCKS_PER_SEC );//time
	
	return 0;
}


























































